/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
* DESCRIPTION :
*
Serial Concurrent Wave Equation - C Version
*
This program implements the concurrent wave equation
* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
# include <stdio.h>
# include <stdlib.h>
# include <math.h>
# include <time.h>
# include <hip/hip_runtime.h>

# define MAXPOINTS 1000000
# define MAXSTEPS 1000000
# define MINPOINTS 20
# define PI 3.14159265
# define DIM_GRID 1
# define BLOCK_SIZE 256



void check_param ( void ) ;
void init_line ( void ) ;
void update ( void ) ;
void printfinal ( void ) ;

int nsteps ,                    /*number of time steps */
    tpoints ,                   /*total points along string */
    rcode ;                     /*generic return code */
float values [ MAXPOINTS +2],   /*values at time t */
      oldval [ MAXPOINTS +2],   /*values at time (t - dt ) */
      newval [ MAXPOINTS +2];   /*values at time ( t + dt ) */

float *d_values, *d_oldval, *d_newval;

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
*                      Checks input values from parameters
* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
void check_param ( void )
{
    char tchar [20];
    /* check number of points , number of iterations */
    while (( tpoints < MINPOINTS ) || ( tpoints > MAXPOINTS ) ) {
        printf ( "Enter number of points along vibrating string [%d-%d]: "
                , MINPOINTS , MAXPOINTS ) ;
        scanf ( "%s" , tchar ) ;
        tpoints = atoi ( tchar ) ;
        if (( tpoints < MINPOINTS ) || ( tpoints > MAXPOINTS ) )
            printf ( "Invalid. Please enter value between %d and %d\n" ,
                    MINPOINTS , MAXPOINTS ) ;
    }
    while (( nsteps < 1) || ( nsteps > MAXSTEPS ) ) {
        printf ( "Enter number of time steps [1-%d]: " , MAXSTEPS ) ;
        scanf ( "%s" , tchar ) ;
        nsteps = atoi ( tchar ) ;
        if (( nsteps < 1) || ( nsteps > MAXSTEPS ) )
            printf ( "Invalid. Please enter value between 1 and %d\n" ,
                    MAXSTEPS ) ;
    }
    printf ( "Using points = %d, steps = %d\n", tpoints, nsteps) ;
}
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
*                                                   Initialize points on line
* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
__global__ void init_line (float* values_d, float* oldvalue_d, int tpoints)
{   
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float fac = 2.0 * PI;
    float x;
    for ( int i = idx; i <= tpoints; i+=stride) {
        x = (float)(i-1)/(tpoints-1);           //might loss some precision.
         values_d[i] = __sinf(fac * x);
         oldvalue_d [i] = values_d[i];
    }
}


/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
*                   Calculate new values using wave equation
* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
*               Update all values along line a specified number of times
* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
__global__ void update(float* values_d, float* oldvalue_d, int tpoints, int nsteps)
{
    int i, j;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    float newval;
    /* Update values for each time step */
    for ( i = 1; i <= nsteps ; i++) {
        /* Update points along line for this time step */
        for ( j = index; j <= tpoints; j+=stride) {
            if (( j == 1) || ( j == tpoints ) )
                newval = 0.0;
            else
                newval  = (1.82)*values_d[j] - oldvalue_d[j];

            oldvalue_d [ j ] = values_d [ j ];   /* Update old values with new values */
            values_d [ j ] = newval;
        }
       
    }
}
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
*                                   Print final results
* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
void printfinal()
{
    int i ;
    for( i = 1; i <= tpoints ; i ++){
        printf("%6.4f ", values[i]);
        if( i %10 == 0)
            printf("\n");
    }
}
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
*                                           Main program
* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
int main (int argc, char *argv[])
{   
    sscanf (argv[1],"%d" ,&tpoints);
    sscanf (argv[2],"%d" ,&nsteps);

    int numBlock = (tpoints+1+BLOCK_SIZE)/BLOCK_SIZE;  
    hipMalloc(&d_values, (MAXPOINTS+2) * sizeof(float));
    hipMalloc(&d_oldval, (MAXPOINTS+2) * sizeof(float));
    check_param();

    printf("Initializing points on the line...\n");
    init_line<<<numBlock, BLOCK_SIZE>>>(d_values, d_oldval, tpoints);
    
    hipMemcpy(values, d_values, (MAXPOINTS+2) * sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 1; i<=tpoints; i++){
        printf("%f\n", *(values+i));
    }
    
    
    printf("Updating all points for all time steps...\n");
    update<<<numBlock, BLOCK_SIZE>>>(d_values, d_oldval, tpoints, nsteps);

    printf("Printing final results...\n");
    hipMemcpy(values, d_values, (MAXPOINTS+2) * sizeof(float), hipMemcpyDeviceToHost);
    printfinal();

    printf("\nDone.\n\n");
    return 0;
}